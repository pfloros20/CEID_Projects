
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>

/* Problem size. */
#define NX 5000
#define NY 5000
#ifndef M_PI
#define M_PI 3.14159
#endif

const unsigned int THREADS_PER_BLOCK = 32;

void init_array(double *x, double *A)
{
	int i, j;

	for (i = 0; i < NX; i++) {
		for (j = 0; j < NY; j++) {
			A[i*NY + j] = ((double) i*(j)) / NX;
		}
	}
	for (i = 0; i < NY; i++) {
		x[i] = i * M_PI;
	}
}

void trans_norm_vector(double* A, double* x, double* y, double* tmp)
{
	int i,j;
	
	for (i= 0; i < NY; i++) {
    	y[i] = 0;
	}
  
	for (i = 0; i < NX; i++) {
      		tmp[i] = 0;

	      	for (j = 0; j < NY; j++) {
			tmp[i] = tmp[i] + A[i*NY + j] * x[j];
		}
		
	      	for (j = 0; j < NY; j++) {
			y[j] = y[j] + A[i*NY + j] * tmp[i];
		}
	}
}

__global__ void trans_norm_vector_kernel(double* A, double* x, double* y, int transpose)
{
  	if(transpose == 0){
		int i = threadIdx.y + blockDim.y * blockIdx.y;
		if(i<NX){
		    y[i] = 0;
		    double tmp = 0;
			for (int j = 0; j < NY; j++)
				tmp = tmp + A[i*NY + j] * x[j];
			y[i] = tmp;
		}
  	}else{
		int j = threadIdx.y + blockDim.y * blockIdx.y;
		if(j<NY){
		   	y[j] = 0;
		   	double tmp = 0;
			for (int i = 0; i < NX; i++)
				tmp = tmp + A[j + i*NY] * x[i];
			y[j] = tmp;
		}
	}
}

__global__ void trans_norm_vector_shared(double* A, double* x, double* y, int transpose)
{
  	if(transpose == 0){
		int i = threadIdx.y + blockDim.y * blockIdx.y;
	    y[i] = 0;
		__shared__ double xs[THREADS_PER_BLOCK];
		double tmp = 0;
		for(int step=0;step<ceil((double)NY/THREADS_PER_BLOCK);step++){
			if(threadIdx.y+step*THREADS_PER_BLOCK>=NY)
				xs[threadIdx.y]=0;
			else
				xs[threadIdx.y]=x[threadIdx.y+step*THREADS_PER_BLOCK];
			__syncthreads();
			for (int j = 0; j < THREADS_PER_BLOCK; j++)
				tmp = tmp + A[i*NY + j + step*THREADS_PER_BLOCK] * xs[j];
			__syncthreads();
		}
		y[i] = tmp;
  	}else{
		int j = threadIdx.y + blockDim.y * blockIdx.y;
		if(j<NY){
		   	y[j] = 0;
		   	double tmp = 0;
			for (int i = 0; i < NX; i++)
				tmp = tmp + A[j + i*NY] * x[i];
			y[j] = tmp;
		}
	}
}

int main(int argc, char *argv[])
{
	//Serial program variables
	double		*y;
	//Host and Device variables for CUDA
	double		*A_h;
	double		*x_h;
	double		*y_h;
	double		*tmp_h;

	double		*A_d;
	double		*x_d;
	double		*y_d;
	double		*tmp_d;
	struct timeval	start, end;

	//Variables for shared memory implementation
	double		*A_s;
	double		*x_s;
	double		*y_s;
	double		*tmp_s;

	A_h = (double*)malloc(NX*NY*sizeof(double));
	x_h = (double*)malloc(NY*sizeof(double));
	y_h = (double*)malloc(NY*sizeof(double));
	tmp_h = (double*)malloc(NX*sizeof(double));
	y = (double*)malloc(NY*sizeof(double));

	init_array(x_h, A_h);


	hipMalloc((void**)&A_d, NX*NY*sizeof(double));
	hipMalloc((void**)&x_d, NY*sizeof(double));
	hipMalloc((void**)&y_d, NY*sizeof(double));
	hipMalloc((void**)&tmp_d, NX*sizeof(double));
	hipMalloc((void**)&A_s, NX*NY*sizeof(double));
	hipMalloc((void**)&x_s, NY*sizeof(double));
	hipMalloc((void**)&y_s, NY*sizeof(double));
	hipMalloc((void**)&tmp_s, NX*sizeof(double));

	fprintf(stdout, "NX: %d NY: %d\n", NX, NY);

	const unsigned int numBlocksInCol= ceil((double)NX/THREADS_PER_BLOCK);
	const unsigned int numBlocksInRow= ceil((double)NY/THREADS_PER_BLOCK);
	dim3 gridDim(1, numBlocksInCol, 1), blockDim(1, THREADS_PER_BLOCK, 1);
	dim3 gridDimT(1, numBlocksInRow, 1);

	//Start of CUDA code
	gettimeofday(&start, NULL);
	hipMemcpy(A_d, A_h, NX*NY*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_h, NY*sizeof(double), hipMemcpyHostToDevice);

	trans_norm_vector_kernel <<< gridDim, blockDim >>>(A_d, x_d, tmp_d, 0);
	trans_norm_vector_kernel <<< gridDimT, blockDim >>>(A_d, tmp_d, y_d, 1);

	hipMemcpy(y_h, y_d , sizeof(double)*NY, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	fprintf(stdout, "GPU Runtime :%0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);



	//Start of CUDA shared code
	gettimeofday(&start, NULL);
	hipMemcpy(A_s, A_h, NX*NY*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(x_s, x_h, NY*sizeof(double), hipMemcpyHostToDevice);

	trans_norm_vector_shared <<< gridDim, blockDim >>>(A_s, x_s, tmp_s, 0);
	trans_norm_vector_shared <<< gridDimT, blockDim >>>(A_s, tmp_s, y_s, 1);

	hipMemcpy(y_h, y_s , sizeof(double)*NY, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	fprintf(stdout, "GPU Shared Runtime :%0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);


	//Start of Serial code
	gettimeofday(&start, NULL);
	trans_norm_vector(A_h, x_h, y, tmp_h);
	gettimeofday(&end, NULL);
	fprintf(stdout, "CPU Runtime :%0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);

	//Error checking CUDA results according to Serial results
	int errors = 0;
	for (int i = 0; i < NY; ++i) {
		double error=fabs((y[i] -y_h[i])/y[i]);
		if(error>pow(10,-14)){
			printf("Error %.20f in (%d)\n",error,i);
			errors++;
		}
	}
	printf("Vector\n\tTotal Results: %d\n\tError count: %d\n",NX*NY,errors);


	free(A_h);
	free(x_h);
	free(y_h);
	free(tmp_h);



	hipFree(A_s); 
	hipFree(x_s); 
	hipFree(y_s); 
	hipFree(tmp_s); 

  	return 0;
}

