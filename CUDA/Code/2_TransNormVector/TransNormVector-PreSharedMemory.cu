
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>

/* Problem size. */
#define NX 4096
#define NY 4096

#ifndef M_PI
#define M_PI 3.14159
#endif

const unsigned int THREADS_PER_BLOCK = 64;

void init_array(double *x, double *A)
{
	int i, j;

	for (i = 0; i < NX; i++) {
		for (j = 0; j < NY; j++) {
			A[i*NY + j] = ((double) i*(j)) / NX;
		}
	}
	for (i = 0; i < NY; i++) {
		x[i] = i * M_PI;
	}
}

__global__ void trans_norm_vector(double* A, double* x, double* y, int transpose)
{
  	if(transpose == 0){
		int i = threadIdx.y + blockDim.y * blockIdx.y;
	    	y[i] = 0;
	    	double tmp = 0;
		    for (int j = 0; j < NY; j++)
				tmp = tmp + A[i*NY + j] * x[j];
			y[i] = tmp;
  	}else{
		int j = threadIdx.y + blockDim.y * blockIdx.y;
	   		y[j] = 0;
	   		double tmp = 0;
			for (int i = 0; i < NX; i++)
				tmp = tmp + A[j + i*NY] * x[i];
			y[j] = tmp;
	}
}

int main(int argc, char *argv[])
{
	double		*A;
	double		*x;
	double		*y;
	double		*tmp;
	struct timeval	gpu_start, gpu_end;

	A = (double*)malloc(NX*NY*sizeof(double));
	x = (double*)malloc(NY*sizeof(double));
	y = (double*)malloc(NY*sizeof(double));
	tmp = (double*)malloc(NX*sizeof(double));

	init_array(x, A);

	double		*A_d;
	double		*x_d;
	double		*y_d;
	double		*tmp_d;

	hipMalloc((void**)&A_d, NX*NY*sizeof(double));
	hipMalloc((void**)&x_d, NY*sizeof(double));
	hipMalloc((void**)&y_d, NY*sizeof(double));
	hipMalloc((void**)&tmp_d, NX*sizeof(double));


	gettimeofday(&gpu_start, NULL);
	hipMemcpy(A_d, A, NX*NY*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(x_d, x, NY*sizeof(double), hipMemcpyHostToDevice);

	const unsigned int numBlocksInCol= ceil((double)NX/THREADS_PER_BLOCK);
	const unsigned int numBlocksInRow= ceil((double)NY/THREADS_PER_BLOCK);
	dim3 gridDim(1, numBlocksInCol, 1), blockDim(1, THREADS_PER_BLOCK, 1);
	dim3 gridDimT(1, numBlocksInRow, 1);

	trans_norm_vector <<< gridDim, blockDim >>>(A_d, x_d, tmp_d, 0);
	trans_norm_vector <<< gridDimT, blockDim >>>(A_d, tmp_d, y_d, 1);
	hipDeviceSynchronize();

	hipMemcpy(y, y_d , sizeof(double)*NY, hipMemcpyDeviceToHost);
	gettimeofday(&gpu_end, NULL);
	fprintf(stdout, "GPU Runtime :%0.6lfs\n", ((gpu_end.tv_sec - gpu_start.tv_sec) * 1000000.0 + (gpu_end.tv_usec - gpu_start.tv_usec)) / 1000000.0);

	//Write results to file
	if(argc == 2)
		if(strcmp(argv[1],"-w") == 0){
			FILE *fp = fopen ("datasetA.txt","w");
		    if (fp == NULL)
		        printf ("File not created.\n");
			
			fwrite(A,sizeof(double),NX*NY,fp) ;
			fclose(fp);
			FILE *fp1 = fopen ("datasetx.txt","w");
			fwrite(x,sizeof(double),NY,fp1) ;
			fclose(fp1);
			FILE *fp2 = fopen ("datasety.txt","w");
			fwrite(y,sizeof(double),NY,fp2) ;
			fclose(fp2);
		}

	free(A);
	free(x);
	free(y);
	free(tmp);

	hipFree(A_d); 
	hipFree(x_d); 
	hipFree(y_d); 
	hipFree(tmp_d); 

  	return 0;
}

