
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>

/* Problem size */
#define NI 5000
#define NJ 5000


#define THREADS_PER_BLOCK_Y 32 
#define THREADS_PER_BLOCK_X 32

void Convolution(double* A, double* B)
{
	int i, j;
	double c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;


	for (i = 1; i < NI - 1; ++i) {
		for (j = 1; j < NJ - 1; ++j) {
			B[i*NJ + j] = c11 * A[(i - 1)*NJ + (j - 1)]  +  c12 * A[(i + 0)*NJ + (j - 1)]  +  c13 * A[(i + 1)*NJ + (j - 1)]
				    + c21 * A[(i - 1)*NJ + (j + 0)]  +  c22 * A[(i + 0)*NJ + (j + 0)]  +  c23 * A[(i + 1)*NJ + (j + 0)] 
				    + c31 * A[(i - 1)*NJ + (j + 1)]  +  c32 * A[(i + 0)*NJ + (j + 1)]  +  c33 * A[(i + 1)*NJ + (j + 1)];
		}
	}
}

__global__ void Convolution_kernel(double* A, double* B)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x+1;
	int i = threadIdx.y + blockDim.y * blockIdx.y+1;
	double c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;
	if(i<NI-1 && j<NJ-1)
		B[i*NJ + j] = c11 * A[(i - 1)*NJ + (j - 1)]  +  c12 * A[(i + 0)*NJ + (j - 1)]  +  c13 * A[(i + 1)*NJ + (j - 1)]
				    + c21 * A[(i - 1)*NJ + (j + 0)]  +  c22 * A[(i + 0)*NJ + (j + 0)]  +  c23 * A[(i + 1)*NJ + (j + 0)] 
				    + c31 * A[(i - 1)*NJ + (j + 1)]  +  c32 * A[(i + 0)*NJ + (j + 1)]  +  c33 * A[(i + 1)*NJ + (j + 1)];
}

__global__ void Convolution_Shared(double* A, double* B)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x+1;
	int i = threadIdx.y + blockDim.y * blockIdx.y+1;
	int tx=threadIdx.y+1;
	int ty=threadIdx.x+1;
	const int sx=THREADS_PER_BLOCK_Y+2;
	const int sy=THREADS_PER_BLOCK_X+2;
	__shared__ double As[sx*sy];
	//if corner element of block matrix, load off block corner neighbor.
	if( tx==1 && ty==1 )
		As[0]=A[(i-1)*NJ+j-1];
	if( tx==sx-2 && ty==sy-2 )
		As[(sx-1)*sy+sy-1]=A[(i+1)*NJ+j+1];
	if( tx==1 && ty==sy-2 )
		As[sy-1]=A[(i-1)*NJ+j+1];
	if( tx==sx-2 && ty==1 )
		As[(sx-1)*sy]=A[(i+1)*NJ+j-1];
	//if border element of block matrix, load off block border neighbor.
	if(tx==1)
		As[ty]=A[(i-1)*NJ+j];
	if(ty==1)
		As[tx*sx]=A[i*NJ+j-1];
	if(tx==sx-2)
		As[(sx-1)*sy+ty]=A[(i+1)*NJ+j];
	if(ty==sy-2)
		As[tx*sy+sy-1]=A[i*NJ+j+1];
	//load cell to shared memory.
	As[tx*sy+ty]=A[i*NJ+j];

	__syncthreads();

	double c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;
	if(i<NI-1 && j<NJ-1)
		B[i*NJ + j] = c11 * As[(tx - 1)*sy+ty - 1]  +  c12 * As[(tx + 0)*sy+ty - 1]  +  c13 * As[(tx + 1)*sy+ty - 1]
					+ c21 * As[(tx - 1)*sy+ty + 0]  +  c22 * As[(tx + 0)*sy+ty + 0]  +  c23 * As[(tx + 1)*sy+ty + 0] 
					+ c31 * As[(tx - 1)*sy+ty + 1]  +  c32 * As[(tx + 0)*sy+ty + 1]  +  c33 * As[(tx + 1)*sy+ty + 1];


}

void init(double* A_h)
{
	int i, j;

	for (i = 0; i < NI; ++i) {
		for (j = 0; j < NJ; ++j) {
			A_h[i*NJ + j] = (double)rand()/RAND_MAX;
        	}
    	}
}


int main(int argc, char *argv[])
{
	//Serial program variables
	double		*B;
	struct timeval	start, end;

	//Host and Device variables for CUDA
	double		*A_h;
	double		*B_h;	
	double		*A_d;
	double		*B_d;
	//Variables for shared memory implementation
	double		*A_s;
	double		*B_s;
	
	//Main Memory allocation
	B = (double*)malloc(NI*NJ*sizeof(double));
	A_h = (double*)malloc(NI*NJ*sizeof(double));
	B_h = (double*)malloc(NI*NJ*sizeof(double));
	//Initialize the array
	init(A_h);

	//GPU Gloval Memory allocation
	hipMalloc((void**)&A_d, NI*NJ*sizeof(double));
	hipMalloc((void**)&B_d, NI*NJ*sizeof(double));
	hipMalloc((void**)&A_s, NI*NJ*sizeof(double));
	hipMalloc((void**)&B_s, NI*NJ*sizeof(double));

	fprintf(stdout, "NI: %d NJ: %d\n", NI, NJ);

	const unsigned int numBlocksInRow = ceil((double)(NJ-2)/THREADS_PER_BLOCK_X);
	const unsigned int numBlocksInCol = ceil((double)(NI-2)/THREADS_PER_BLOCK_Y);
	dim3 gridDim(numBlocksInRow, numBlocksInCol, 1), blockDim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);

	//Start of CUDA code

	gettimeofday(&start, NULL);
	hipMemcpy(A_d, A_h, NI*NJ*sizeof(double), hipMemcpyHostToDevice);

	Convolution_kernel <<< gridDim, blockDim >>>(A_d, B_d);
	hipMemcpy(B_h, B_d , sizeof(double)*NI*NJ, hipMemcpyDeviceToHost); 
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);


	//Start of CUDA shared code
	gettimeofday(&start, NULL);
	hipMemcpy(A_s, A_h, NI*NJ*sizeof(double), hipMemcpyHostToDevice);

	Convolution_Shared <<< gridDim, blockDim >>>(A_s, B_s);
	hipMemcpy(B_h, B_s , sizeof(double)*NI*NJ, hipMemcpyDeviceToHost); 
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	fprintf(stdout, "GPU Shared Runtime: %0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);



	//Start of Serial code
	gettimeofday(&start, NULL);
	Convolution(A_h, B);
	gettimeofday(&end, NULL);
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", ((end.tv_sec - start.tv_sec) * 1000000.0 + (end.tv_usec - start.tv_usec)) / 1000000.0);

	//Error checking CUDA results according to Serial results
	int errors = 0;
	for (int i = 0; i < NI; ++i) {
		for (int j = 0; j < NJ; ++j) {
			double error=fabs((B[i*NJ + j] -B_h[i*NJ+j]));
			if(error>pow(10,-14)){
				printf("Error %.20f in (%d,%d)\n",error,i,j);
				errors++;
			}
		}
	}
	printf("Matrix\n\tTotal Results: %d\n\tError count: %d\n",NI*NJ,errors);
	

	free(A_h);
	free(B_h);
	free(B);

	hipFree(A_d); 
	hipFree(B_d); 
	hipFree(A_s); 
	hipFree(B_s); 
	
	return 0;
}

