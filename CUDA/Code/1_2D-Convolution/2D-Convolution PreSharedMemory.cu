
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>

/* Problem size */
#define NI 5000
#define NJ 5000


const unsigned int THREADS_PER_BLOCK_Y = 32; 
const unsigned int THREADS_PER_BLOCK_X = 32; 

__global__ void Convolution(double* A, double* B, double* C)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x+1;
	int i = threadIdx.y + blockDim.y * blockIdx.y+1;

	if(i<NI-1 && j<NJ-1)
		B[i*NJ + j] = C[0] * A[(i - 1)*NJ + (j - 1)]  +  C[3] * A[(i + 0)*NJ + (j - 1)]  +  C[6] * A[(i + 1)*NJ + (j - 1)]
				    + C[1] * A[(i - 1)*NJ + (j + 0)]  +  C[4] * A[(i + 0)*NJ + (j + 0)]  +  C[7] * A[(i + 1)*NJ + (j + 0)] 
				    + C[2] * A[(i - 1)*NJ + (j + 1)]  +  C[5] * A[(i + 0)*NJ + (j + 1)]  +  C[8] * A[(i + 1)*NJ + (j + 1)];

}

void init(double* A_h)
{
	int i, j;

	for (i = 0; i < NI; ++i) {
		for (j = 0; j < NJ; ++j) {
			A_h[i*NJ + j] = (double)rand()/RAND_MAX;
        	}
    	}
}

int main(int argc, char *argv[])
{
	double		*A_h;
	double		*B_h;
	struct timeval	gpu_start, gpu_end;

	
	A_h = (double*)malloc(NI*NJ*sizeof(double));
	B_h = (double*)malloc(NI*NJ*sizeof(double));
	double C_h[]= {+0.2, +0.5, -0.8, -0.3, +0.6, -0.9, +0.4, +0.7, +0.10};
	//initialize the array
	init(A_h);
	
	double		*A_d;
	double		*B_d;
	double		*C_d;

	hipMalloc((void**)&A_d, NI*NJ*sizeof(double));
	hipMalloc((void**)&B_d, NI*NJ*sizeof(double));
	hipMalloc((void**)&C_d, 9*sizeof(double));
	gettimeofday(&gpu_start, NULL);
	hipMemcpy(A_d, A_h, NI*NJ*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(C_d, C_h, 9*sizeof(double), hipMemcpyHostToDevice);
	
	//const unsigned int numBlocks = (NI-2)*(NJ-2)/THREADS_PER_BLOCK + 1;
	const unsigned int numBlocksInRow = ceil((double)(NJ-2)/THREADS_PER_BLOCK_X);
	const unsigned int numBlocksInCol = ceil((double)(NI-2)/THREADS_PER_BLOCK_Y);
	dim3 gridDim(numBlocksInRow, numBlocksInCol, 1), blockDim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);

	Convolution <<< gridDim, blockDim >>>(A_d, B_d, C_d);
	hipDeviceSynchronize();
	hipMemcpy(B_h, B_d , sizeof(double)*NI*NJ, hipMemcpyDeviceToHost); 
	gettimeofday(&gpu_end, NULL);
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", ((gpu_end.tv_sec - gpu_start.tv_sec) * 1000000.0 + (gpu_end.tv_usec - gpu_start.tv_usec)) / 1000000.0);

	//Write results to file
	if(argc == 2)
		if(strcmp(argv[1],"-w") == 0){
			FILE *fp = fopen ("datasetA.txt","w");
		    if (fp == NULL)
		        printf ("File not created.\n");
			
			fwrite(A_h,sizeof(double),NI*NJ,fp) ;
			fclose(fp);
			FILE *fp1 = fopen ("datasetB.txt","w");
			fwrite(B_h,sizeof(double),NI*NJ,fp1) ;
			fclose(fp1);
		}
		        	

	free(A_h);
	free(B_h);

	hipFree(A_d); 
	hipFree(B_d); 
	
	return 0;
}

