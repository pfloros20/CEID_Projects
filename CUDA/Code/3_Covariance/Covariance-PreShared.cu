
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>

/* Problem size */
#define M 1024
#define N 1024

#define FLOAT_N 3214212.01


const unsigned int THREADS_PER_BLOCK = 32;

void init_arrays(double* data)
{
	int i, j;

	for (i = 0; i < M; i++) {
		for (j = 0; j < N; j++) {
			data[i*N + j] = ((double) i*j) / M;
		}
	}
}

void covariance(double* data, double* symmat, double* mean)
{
	int	i, j, j1,j2;

  	/* Determine mean of column vectors of input data matrix */
	for (j = 0; j < M; j++) {
		mean[j] = 0.0;
		for (i = 0; i < N; i++) {
        		mean[j] += data[i*M + j];
		}
		mean[j] /= FLOAT_N;
	}

  	/* Center the column vectors. */
	for (i = 0; i < N; i++) {
		for (j = 0; j < M; j++) {
			data[i*M + j] -= mean[j];
		}
	}

  	/* Calculate the m * m covariance matrix. */
	for (j1 = 0; j1 < M; j1++) {
		for (j2 = j1; j2 < M; j2++) {
	       	symmat[j1*M + j2] = 0.0;
			for (i = 0; i < N; i++) {
				symmat[j1*M + j2] += data[i*M + j1] * data[i*M + j2];
			}
        	symmat[j2*M + j1] = symmat[j1*M + j2];
      	}
	}
}

__global__ void covariance_kernel(double* data, double* symmat, double* mean)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i,j1;
  	/* Determine mean of column vectors of input data matrix */
	mean[j] = 0.0;
	double temp = 0;
	for (i = 0; i < N; i++) {
    	temp += data[i*M + j];
	}
	double loc_mean =temp/FLOAT_N;
	mean[j] = loc_mean;
  	/* Center the column vectors. */
	for (i = 0; i < N; i++) {
		data[i*M + j] -= loc_mean;
	}
  	/* Calculate the m * m covariance matrix. */
	for (j1 = j; j1 < M; j1++) {
	    symmat[j*M + j1] = 0.0;
  		temp = 0;
		for (i = 0; i < N; i++) {
			 temp+= data[i*M + j] * data[i*M + j1];
		}
		symmat[j*M + j1] = temp;
        symmat[j1*M + j] = temp;
    }
}

int main(int argc, char *argv[])
{
	//Serial program variables
	double		*data;
	double		*symmat;
	double		*mean;
	struct timeval	cpu_start, cpu_end;
	//Host and Device variables for CUDA
	double		*data_h;
	double		*symmat_h;
	double		*mean_h;
	double		*data_d;
	double		*symmat_d;
	double		*mean_d;
	struct timeval	gpu_start, gpu_end;

	//Main Memory allocation
	data = (double*)malloc(M*N*sizeof(double));
	symmat = (double*)malloc(M*M*sizeof(double));
	mean = (double*)malloc(M*sizeof(double));

	data_h = (double*)malloc(M*N*sizeof(double));
	symmat_h = (double*)malloc(M*M*sizeof(double));
	mean_h = (double*)malloc(M*sizeof(double));

	//Initialize data
	init_arrays(data);
	init_arrays(data_h);

	//GPU Gloval Memory allocation
	hipMalloc((void**)&data_d, M*N*sizeof(double));
	hipMalloc((void**)&symmat_d, M*M*sizeof(double));
	hipMalloc((void**)&mean_d, M*sizeof(double));

	//Start of CUDA code
	gettimeofday(&gpu_start, NULL);
	hipMemcpy(data_d, data_h, M*N*sizeof(double), hipMemcpyHostToDevice);

	//const unsigned int numBlocksInCol= ceil((double)M/THREADS_PER_BLOCK);
	const unsigned int numBlocksInRow= ceil((double)N/THREADS_PER_BLOCK);
	dim3 gridDim(numBlocksInRow, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);

	//Kernel Call lul
	covariance_kernel <<< gridDim, blockDim >>>(data_d, symmat_d, mean_d);

	hipMemcpy(symmat_h, symmat_d , M*M*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&gpu_end, NULL);
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", ((gpu_end.tv_sec - gpu_start.tv_sec) * 1000000.0 + (gpu_end.tv_usec - gpu_start.tv_usec)) / 1000000.0);

	//Start of Serial code
	gettimeofday(&cpu_start, NULL);
	covariance(data, symmat, mean);
	gettimeofday(&cpu_end, NULL);
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", ((cpu_end.tv_sec - cpu_start.tv_sec) * 1000000.0 + (cpu_end.tv_usec - cpu_start.tv_usec)) / 1000000.0);

	//Error checking CUDA results according to Serial results
	int errors = 0;
	for (int i = 0; i < M; ++i) {
		for (int j = 0; j < M; ++j) {
			double error=fabs((symmat[i*M + j] -symmat_h[i*M+j])/symmat_h[i*M+j]);
			if(error>0.000000000000001){
				//printf("Error %.20f in (%d,%d)\n",error,i,j);
				//printf("Value %.20f in (%d,%d)\n",symmat_h[i*M+j],i,j);
				errors++;
			}
		}
	}
	printf("Symmat\n\tTotal Results: %d\n\tError count: %d\n",M*M,errors);



	// cudaMemcpy(data_h, data_d , M*N*sizeof(double), cudaMemcpyDeviceToHost);
	// errors = 0;
	// for (int i = 0; i < M; ++i) {
	// 	for (int j = 0; j < N; ++j) {
	// 		double error=fabs(data[i*N + j] -data_h[i*N+j]);
	// 		if(error>0.000000000000001){
	// 			//printf("Error %.20f in (%d,%d)\n",error,i,j);
	// 			errors++;
	// 		}
	// 	}
	// }
	// printf("Data\n\tTotal Results: %d\n\tError count: %d\n",M*M,errors);


	//Free allocated memory
	free(data);
	free(symmat);
	free(mean);
	free(data_h);
	free(symmat_h);
	free(mean_h);

	hipFree(data_d); 
	hipFree(symmat_d); 
	hipFree(mean_d);

  	return 0;
}

